#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#ifdef USE_ROCM
#include "quantization/fp8/amd/quant_utils.cuh"
#else
#include "quantization/fp8/nvidia/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
  typedef __hip_bfloat16 __hip_bfloat16;
#endif

void swap_blocks(
  torch::Tensor& src,
  torch::Tensor& dst,
  const torch::Tensor& block_mapping) {
  torch::Device src_device = src.device();
  torch::Device dst_device = dst.device();
  hipMemcpyKind memcpy_type;
  if (src_device.is_cuda() && dst_device.is_cuda()) {
    TORCH_CHECK(
      src_device.index() == dst_device.index(),
      "src and dst must be on the same GPU");
    memcpy_type = hipMemcpyDeviceToDevice;
  } else if (src_device.is_cuda() && dst_device.is_cpu()) {
    memcpy_type = hipMemcpyDeviceToHost;
  } else if (src_device.is_cpu() && dst_device.is_cuda()) {
    memcpy_type = hipMemcpyHostToDevice;
  } else {
    TORCH_CHECK(false, "Invalid device combination");
  }

  // NOTE(youkaichao): keep in mind that `block_mapping` should be 
  // a cpu tensor, otherwise every `item` call will require a gpu-cpu
  // synchronization.
  TORCH_CHECK(block_mapping.device().is_cpu(), "block_mapping must be on CPU");

  char *src_ptr = static_cast<char*>(src.data_ptr());
  char *dst_ptr = static_cast<char*>(dst.data_ptr());

  const int64_t block_size_in_bytes = src.element_size() * src[0].numel();
  const at::cuda::OptionalCUDAGuard device_guard(src_device.is_cuda() ? src_device : dst_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  // NOTE(woosuk): This can be slow if the number of blocks is large.
  const int64_t num_blocks = block_mapping.size(0);
  for (size_t i = 0; i < num_blocks; i++) {
    int64_t src_block_number = block_mapping[i][0].item<int64_t>();
    int64_t dst_block_number = block_mapping[i][1].item<int64_t>();
    int64_t src_offset = src_block_number * block_size_in_bytes;
    int64_t dst_offset = dst_block_number * block_size_in_bytes;
    hipMemcpyAsync(
      dst_ptr + dst_offset,
      src_ptr + src_offset,
      block_size_in_bytes,
      memcpy_type,
      stream);
  }
}

// void sparse_cache_copy(
//   torch::Tensor& src,
//   torch::Tensor& dst,
//   const torch::Tensor& block_mapping,
//   const std::vector<torch::Tensor>& selection) {
//   torch::Device src_device = src.device();
//   torch::Device dst_device = dst.device();
//   TORCH_CHECK(
//       src_device.index() == dst_device.index(),
//       "src and dst must be on the same GPU");
//   hipMemcpyKind memcpy_type = udaMemcpyDeviceToDevice;

//   TORCH_CHECK(block_mapping.device().is_cpu(), "block_mapping must be on CPU");
//   char *src_ptr = static_cast<char*>(src.data_ptr());
//   char *dst_ptr = static_cast<char*>(dst.data_ptr());

// }


// namespace vllm {

// // Grid: (num_layers, num_pairs)
// template<typename scalar_t>
// __global__ void sparse_cache_copy_kernel(
//   int64_t* key_cache_ptrs,
//   int64_t* value_cache_ptrs,
//   const int64_t* __restrict__ block_mapping,
//   const int64_t* __restrict__ selection,
//   const int numel_per_block) {
//   const int layer_idx = blockIdx.x;
//   const int pair_idx = blockIdx.y;

//   scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
//   scalar_t* value_cache = reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);
//   int64_t src_block_number = block_mapping[2 * pair_idx];
//   int64_t dst_block_number = block_mapping[2 * pair_idx + 1];

//   const int64_t src_block_offset = src_block_number * numel_per_block;
//   const int64_t dst_block_offset = dst_block_number * numel_per_block;
//   for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
//     int64_t src_offset = src_block_offset + i;
//     int64_t dst_offset = dst_block_offset + i;
//     key_cache[dst_offset] = key_cache[src_offset];
//   }
//   for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
//     int64_t src_offset = src_block_offset + i;
//     int64_t dst_offset = dst_block_offset + i;
//     value_cache[dst_offset] = value_cache[src_offset];
//   }
// }

// } // namespace vllm


// void sparse_cache_copy(
//   const std::vector<torch::Tensor>& key_caches,
//   std::vector<torch::Tensor>& value_caches,
//   const torch::Tensor& block_mapping,
//   const std::vector<torch::Tensor>& selection) {
//   int num_layers = key_caches.size();
//   TORCH_CHECK(num_layers == value_caches.size());
//   if (num_layers == 0) {
//     return;
//   }
//   torch::Device cache_device = key_caches[0].device();
//   TORCH_CHECK(cache_device.is_cuda());

//   // Create data structures for the kernel.
//   // Create an array of pointers to the key and value caches.
//   int64_t key_cache_ptrs[num_layers];
//   int64_t value_cache_ptrs[num_layers];
//   for (int layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
//     key_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(key_caches[layer_idx].data_ptr());
//     value_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(value_caches[layer_idx].data_ptr());
//   }

//   // block_mapping is a 2D tensor with shape (num_pairs, 2).
//   int num_pairs = block_mapping.size(0);

//   // Move the data structures to the GPU.
//   // NOTE: This synchronizes the CPU and GPU.
//   torch::Tensor key_cache_ptrs_tensor = torch::from_blob(
//     key_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
//   torch::Tensor value_cache_ptrs_tensor = torch::from_blob(
//     value_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);

//   // Launch the kernel.
//   const int numel_per_block = key_caches[0][0].numel();
//   dim3 grid(num_layers, num_pairs);
//   dim3 block(std::min(1024, numel_per_block));
//   const at::cuda::OptionalCUDAGuard device_guard(cache_device);
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
//     key_caches[0].scalar_type(), "sparse_cache_copy_kernel", ([&] {
//       vllm::sparse_cache_copy_kernel<scalar_t><<<grid, block, 0, stream>>>(
//         key_cache_ptrs_tensor.data_ptr<int64_t>(),
//         value_cache_ptrs_tensor.data_ptr<int64_t>(),
//         block_mapping.data_ptr<int64_t>(),
//         selection.data_ptr<int64_t>(),
//         numel_per_block);
//     }));
// }

// namespace vllm {

// // Grid: (num_layers, 1)
// template <typename scalar_t>
// __global__ void sparse_cache_copy_kernel(
//   const scalar_t* __restrict__ src,
//   scalar_t* __restrict__ target,
//   const int* __restrict__ selection,
//   int64_t* __restrict__ target_indices,
//   const int64_t num_elements) {
  
//   const int layer_idx = blockIdx.x;
//   const int idx = threadIdx.x + blockDim.x * blockIdx.y;

//   for (int64_t i = idx; i < num_elements; i += blockDim.x * gridDim.y) {
//     if (selection[layer_idx * num_elements + i] == 1) {
//       unsigned long long int* address = (unsigned long long int*)&target_indices[layer_idx];
//       const int64_t target_idx = atomicAdd(address, 1ULL);
//       //const int64_t target_idx = atomicAdd(&target_indices[layer_idx], 1);
//       target[layer_idx * num_elements + target_idx] = src[layer_idx * num_elements + i];
//     }
//   }
// }

// } // namespace vllm


// void sparse_cache_copy(
//   const std::vector<torch::Tensor>& src,
//   std::vector<torch::Tensor>& target,
//   // const torch::Tensor& block_mapping,
//   const std::vector<torch::Tensor>& selection) {

//   TORCH_CHECK(src.size() == target.size(), "src and target must have the same size");
//   // TORCH_CHECK(src.size() == selection.size(), "src and selection must have the same size");

//   const int num_layers = src.size();
//   if (num_layers == 0) {
//     return;
//   }

//   torch::Device device = src[0].device();
//   TORCH_CHECK(device.is_cuda());

//   // Create data structures for the kernel
//   // ?? why num_layers
//   std::vector<int64_t> src_ptrs(num_layers);
//   std::vector<int64_t> target_ptrs(num_layers);
//   std::vector<int64_t> selection_ptrs(num_layers);
//   for (int layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
//     src_ptrs[layer_idx] = reinterpret_cast<int64_t>(src[layer_idx].data_ptr());
//     target_ptrs[layer_idx] = reinterpret_cast<int64_t>(target[layer_idx].data_ptr());
//     selection_ptrs[layer_idx] = reinterpret_cast<int64_t>(selection[layer_idx].data_ptr());
//   }

//   torch::Tensor src_ptrs_tensor = torch::from_blob(
//     src_ptrs.data(), {num_layers}, torch::kInt64).to(device);
//   torch::Tensor target_ptrs_tensor = torch::from_blob(
//     target_ptrs.data(), {num_layers}, torch::kInt64).to(device);
//   torch::Tensor selection_ptrs_tensor = torch::from_blob(
//     selection_ptrs.data(), {num_layers}, torch::kInt64).to(device);

//   // Allocate memory for target_indices
//   torch::Tensor target_indices = torch::zeros({num_layers}, torch::dtype(torch::kInt64).device(device));

//   // Launch the kernel
//   // ??
//   const int64_t num_elements = src[0].numel();
//   const int num_threads = 1024;
//   dim3 block(num_threads);
//   dim3 grid(num_layers, (num_elements + num_threads - 1) / num_threads);

//   const at::cuda::OptionalCUDAGuard device_guard(device);
//   const hipStream_t stream = at::cuda::getCurrentCUDAStream();
//   VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
//     src[0].scalar_type(), "sparse_cache_copy_kernel", ([&] {
//       vllm::sparse_cache_copy_kernel<scalar_t><<<grid, block, 0, stream>>>(
//         src_ptrs_tensor.data_ptr<scalar_t>(),
//         target_ptrs_tensor.data_ptr<scalar_t>(),
//         selection_ptrs_tensor.data_ptr<int>(),
//         target_indices.data_ptr<int64_t>(),
//         num_elements);
//     }));
// }

namespace vllm {

// Grid: (num_layers, num_pairs)
template<typename scalar_t>
__global__ void copy_blocks_kernel(
  int64_t* key_cache_ptrs,
  int64_t* value_cache_ptrs,
  const int64_t* __restrict__ block_mapping,
  const int numel_per_block) {
  const int layer_idx = blockIdx.x;
  const int pair_idx = blockIdx.y;

  scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
  scalar_t* value_cache = reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);
  int64_t src_block_number = block_mapping[2 * pair_idx];
  int64_t dst_block_number = block_mapping[2 * pair_idx + 1];

  const int64_t src_block_offset = src_block_number * numel_per_block;
  const int64_t dst_block_offset = dst_block_number * numel_per_block;
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    key_cache[dst_offset] = key_cache[src_offset];
  }
  for (int i = threadIdx.x; i < numel_per_block; i += blockDim.x) {
    int64_t src_offset = src_block_offset + i;
    int64_t dst_offset = dst_block_offset + i;
    value_cache[dst_offset] = value_cache[src_offset];
  }
}

} // namespace vllm

void copy_blocks(
  std::vector<torch::Tensor>& key_caches,
  std::vector<torch::Tensor>& value_caches,
  const torch::Tensor& block_mapping) {
  int num_layers = key_caches.size();
  TORCH_CHECK(num_layers == value_caches.size());
  if (num_layers == 0) {
    return;
  }
  torch::Device cache_device = key_caches[0].device();
  TORCH_CHECK(cache_device.is_cuda());

  // Create data structures for the kernel.
  // Create an array of pointers to the key and value caches.
  int64_t key_cache_ptrs[num_layers];
  int64_t value_cache_ptrs[num_layers];
  for (int layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
    key_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(key_caches[layer_idx].data_ptr());
    value_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(value_caches[layer_idx].data_ptr());
  }

  // block_mapping is a 2D tensor with shape (num_pairs, 2).
  int num_pairs = block_mapping.size(0);

  // Move the data structures to the GPU.
  // NOTE: This synchronizes the CPU and GPU.
  torch::Tensor key_cache_ptrs_tensor = torch::from_blob(
    key_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
  torch::Tensor value_cache_ptrs_tensor = torch::from_blob(
    value_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);

  // Launch the kernel.
  const int numel_per_block = key_caches[0][0].numel();
  dim3 grid(num_layers, num_pairs);
  dim3 block(std::min(1024, numel_per_block));
  const at::cuda::OptionalCUDAGuard device_guard(cache_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
    key_caches[0].scalar_type(), "copy_blocks_kernel", ([&] {
      vllm::copy_blocks_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key_cache_ptrs_tensor.data_ptr<int64_t>(),
        value_cache_ptrs_tensor.data_ptr<int64_t>(),
        block_mapping.data_ptr<int64_t>(),
        numel_per_block);
    }));
}

namespace vllm {

// Grid: (num_selected_layers, num_src_blocks)
template<typename scalar_t>
__global__ void sparse_cache_copy_kernel(
  int64_t* key_cache_ptrs,
  int64_t* value_cache_ptrs,
  const int64_t* __restrict__ block_mapping_src,
  const int64_t* __restrict__ block_mapping_dst,
  const int numel_per_block,
  const int* __restrict__ selection_index_src,
  const int* __restrict__ selection_index_dst,
  const int num_selected_pairs) {
  const int pair_idx = blockIdx.x;
  const int src_block_idx = blockIdx.y;

  if (pair_idx >= num_selected_pairs) {
    return;
  }

  // Get the layer index from selection_index
  const int layer_idx = selection_index_src[pair_idx];
  const int dst_idx = selection_index_dst[pair_idx];

  scalar_t* key_cache = reinterpret_cast<scalar_t*>(key_cache_ptrs[layer_idx]);
  scalar_t* value_cache = reinterpret_cast<scalar_t*>(value_cache_ptrs[layer_idx]);

  // printf("numel %d", numel_per_block); // 12288??
  int block_mapping_src_idx = src_block_idx/numel_per_block;
  int block_mapping_dst_idx = dst_idx/numel_per_block;
  int64_t src_block_number = block_mapping_src[block_mapping_src_idx]; //src_block_idx
  int64_t dst_block_number = block_mapping_dst[block_mapping_dst_idx]; //dst_idx % num_selected_pairs
  const int64_t src_block_offset = src_block_number * numel_per_block;
  const int64_t dst_block_offset = dst_block_number * numel_per_block;

  for (int elem_idx = threadIdx.x; elem_idx < numel_per_block; elem_idx += blockDim.x) {
    int64_t src_offset = src_block_offset + elem_idx;
    int64_t dst_offset = dst_block_offset + elem_idx;

    key_cache[dst_offset] = key_cache[src_offset];
    value_cache[dst_offset] = value_cache[src_offset];
  }
}

} // namespace vllm

void sparse_cache_copy(
  std::vector<torch::Tensor>& key_caches,
  std::vector<torch::Tensor>& value_caches,
  std::vector<int64_t>& block_mapping_src,
  std::vector<int64_t>& block_mapping_dst,
  const torch::Tensor& selection) {
  int num_layers = key_caches.size();

  int block_mapping_src_number = static_cast<int64_t>(block_mapping_src.size());

  int block_mapping_dst_number = static_cast<int64_t>(block_mapping_dst.size());
  printf("This is sparse copy %d, %d, %d\n",num_layers, value_caches.size(), selection.size(0));
  TORCH_CHECK(num_layers == value_caches.size());
  TORCH_CHECK(selection.size(0) == num_layers * 16 * block_mapping_src_number);
  if (num_layers == 0) {
    return;
  }
  torch::Device cache_device = key_caches[0].device();
  TORCH_CHECK(cache_device.is_cuda());
  //TORCH_CHECK(selection.device().is_cuda());

  // Create data structures for the kernel.
  int64_t key_cache_ptrs[num_layers];
  int64_t value_cache_ptrs[num_layers];
  for (int layer_idx = 0; layer_idx < num_layers; ++layer_idx) {
    key_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(key_caches[layer_idx].data_ptr());
    value_cache_ptrs[layer_idx] = reinterpret_cast<int64_t>(value_caches[layer_idx].data_ptr());
  }

  // Create selection_index tensor based on selection
  std::vector<int> selection_index_src;
  std::vector<int> selection_index_dst;
  int dst_idx = 0;
  for (int src_idx = 0; src_idx < selection.size(0); ++src_idx) {
    if (selection[src_idx].item<int>() == 1) {
      selection_index_src.push_back(src_idx);
      selection_index_dst.push_back(dst_idx++);
    }
  }
  int num_selected_pairs = selection_index_src.size();
  printf("num_selected_pairs %d\n", num_selected_pairs);
  printf("selection_index_src first item %d\n", selection_index_src[0]);
  printf("selection_index_src last item %d\n", selection_index_src[selection_index_src.size()-1]);
  printf("selection_index_dst first item %d\n", selection_index_dst[0]);
  printf("selection_index_dst last item %d\n", selection_index_dst[selection_index_dst.size()-1]);

  int numel_per_block = key_caches[0][0].numel();


  // Move the data structures to the GPU.
  torch::Tensor key_cache_ptrs_tensor = torch::from_blob(
    key_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
  torch::Tensor value_cache_ptrs_tensor = torch::from_blob(
    value_cache_ptrs, {num_layers}, torch::kInt64).to(cache_device);
  torch::Tensor selection_index_src_tensor = torch::from_blob(
    selection_index_src.data(), {num_selected_pairs}, torch::kInt32).to(cache_device);
  torch::Tensor selection_index_dst_tensor = torch::from_blob(
    selection_index_dst.data(), {num_selected_pairs}, torch::kInt32).to(cache_device);
  torch::Tensor block_mapping_src_tensor = torch::from_blob(
    block_mapping_src.data(), {block_mapping_src_number}, torch::kInt64).to(cache_device);
  torch::Tensor block_mapping_dst_tensor = torch::from_blob(
    block_mapping_dst.data(), {block_mapping_dst_number}, torch::kInt64).to(cache_device);


  // Launch the kernel.
  dim3 grid(num_selected_pairs, block_mapping_src.size());
  dim3 block(std::min(1024, numel_per_block));
  const at::cuda::OptionalCUDAGuard device_guard(cache_device);
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_AND_BYTE_TYPES(
    key_caches[0].scalar_type(), "sparse_cache_copy_kernel", ([&] {
      vllm::sparse_cache_copy_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key_cache_ptrs_tensor.data_ptr<int64_t>(),
        value_cache_ptrs_tensor.data_ptr<int64_t>(),
        block_mapping_src_tensor.data_ptr<int64_t>(),
        block_mapping_dst_tensor.data_ptr<int64_t>(),
        numel_per_block,
        selection_index_src_tensor.data_ptr<int>(),
        selection_index_dst_tensor.data_ptr<int>(),
        num_selected_pairs);
    }));
}


namespace vllm {

template<typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
__global__ void reshape_and_cache_kernel(
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  cache_t* __restrict__ key_cache,            // [num_blocks, num_heads, head_size/x, block_size, x]
  cache_t* __restrict__ value_cache,          // [num_blocks, num_heads, head_size, block_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size,
  const int x,
  const float kv_scale) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  if (slot_idx < 0) {
    // Padding token that should be ignored.
    return;
  }

  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;

    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int x_idx = head_offset / x;
    const int x_offset = head_offset % x;

    const int64_t tgt_key_idx = block_idx * num_heads * (head_size / x) * block_size * x
                                + head_idx * (head_size / x) * block_size * x
                                + x_idx * block_size * x
                                + block_offset * x
                                + x_offset;
    const int64_t tgt_value_idx = block_idx * num_heads * head_size * block_size
                                  + head_idx * head_size * block_size
                                  + head_offset * block_size
                                  + block_offset;
    scalar_t tgt_key = key[src_key_idx];
    scalar_t tgt_value = value[src_value_idx];
    if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
      key_cache[tgt_key_idx] = tgt_key;
      value_cache[tgt_value_idx] = tgt_value;
    } else {
      key_cache[tgt_key_idx] = fp8::scaled_convert<cache_t, scalar_t, kv_dt>(tgt_key, kv_scale);
      value_cache[tgt_value_idx] = fp8::scaled_convert<cache_t, scalar_t, kv_dt>(tgt_value, kv_scale);
    }
  }
}

template<typename scalar_t>
__global__ void reshape_and_cache_flash_kernel(
  const scalar_t* __restrict__ key,           // [num_tokens, num_heads, head_size]
  const scalar_t* __restrict__ value,         // [num_tokens, num_heads, head_size]
  scalar_t* __restrict__ k_cache,             // [num_blocks, block_size, num_heads, head_size]
  scalar_t* __restrict__ v_cache,             // [num_blocks, block_size, num_heads, head_size]
  const int64_t* __restrict__ slot_mapping,   // [num_tokens]
  const int block_stride,
  const int key_stride,
  const int value_stride,
  const int num_heads,
  const int head_size,
  const int block_size) {
  const int64_t token_idx = blockIdx.x;
  const int64_t slot_idx = slot_mapping[token_idx];
  // NOTE: slot_idx can be -1 if the token is padded
  if (slot_idx < 0) {
    return;
  }
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;
  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_key_idx = token_idx * key_stride + i;
    const int64_t src_value_idx = token_idx * value_stride + i;
    const int head_idx = i / head_size;
    const int head_offset = i % head_size;
    const int64_t tgt_value_idx = block_idx * block_stride
                              + block_offset * num_heads * head_size
                              + head_idx * head_size
                              + head_offset;
    k_cache[tgt_value_idx] = key[src_key_idx];
    v_cache[tgt_value_idx] = value[src_value_idx];
  }
}
} // namespace vllm

// KV_T is the stored data type of kv-cache.
// CACHE_T is the data type of key and value tensors.
// KV_DTYPE is the real data type of kv-cache.
#define CALL_RESHAPE_AND_CACHE(KV_T, CACHE_T, KV_DTYPE)                                     \
  vllm::reshape_and_cache_kernel<KV_T, CACHE_T, KV_DTYPE><<<grid, block, 0, stream>>>(      \
    reinterpret_cast<KV_T*>(key.data_ptr()),                                                \
    reinterpret_cast<KV_T*>(value.data_ptr()),                                              \
    reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),                                       \
    reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),                                     \
    slot_mapping.data_ptr<int64_t>(),                                                       \
    key_stride,                                                                             \
    value_stride,                                                                           \
    num_heads,                                                                              \
    head_size,                                                                              \
    block_size,                                                                             \
    x,                                                                                      \
    kv_scale);

void reshape_and_cache(
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
  torch::Tensor& value_cache,   // [num_blocks, num_heads, head_size, block_size]
  torch::Tensor& slot_mapping,  // [num_tokens]
  const std::string& kv_cache_dtype,
  const float kv_scale)
{
  int num_tokens = key.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = key_cache.size(3);
  int x = key_cache.size(4);

  int key_stride = key.stride(0);
  int value_stride = value.stride(0);

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  DISPATCH_BY_KV_CACHE_DTYPE(key.dtype(), kv_cache_dtype, CALL_RESHAPE_AND_CACHE)
}

void reshape_and_cache_flash(
  torch::Tensor& key,           // [num_tokens, num_heads, head_size]
  torch::Tensor& value,         // [num_tokens, num_heads, head_size]
  torch::Tensor& k_cache,       // [num_blocks, block_size, num_heads, head_size]
  torch::Tensor& v_cache,       // [num_blocks, block_size, num_heads, head_size]
  torch::Tensor& slot_mapping,  // [num_tokens]
  const std::string& kv_cache_dtype)
{
  // FIXME: only support auto datatype, does not support fp8
  if (kv_cache_dtype != "auto") {
    TORCH_CHECK(false, "Unsupported data type of kv cache: ", kv_cache_dtype);
  }
  int num_tokens = key.size(0);
  int num_heads = key.size(1);
  int head_size = key.size(2);
  int block_size = k_cache.size(1);

  int key_stride = key.stride(0);
  int value_stride = value.stride(0);
  int block_stride = k_cache.stride(0);
  TORCH_CHECK(k_cache.stride(0) == v_cache.stride(0));

  dim3 grid(num_tokens);
  dim3 block(std::min(num_heads * head_size, 512));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(key));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
    key.scalar_type(),
    "reshape_and_cache_flash",
    [&] {
      vllm::reshape_and_cache_flash_kernel<scalar_t><<<grid, block, 0, stream>>>(
        key.data_ptr<scalar_t>(),
        value.data_ptr<scalar_t>(),
        k_cache.data_ptr<scalar_t>(),
        v_cache.data_ptr<scalar_t>(),
        slot_mapping.data_ptr<int64_t>(),
        block_stride,
        key_stride,
        value_stride,
        num_heads,
        head_size,
        block_size);
    });
}

namespace vllm {

template<typename Tout, typename Tin, Fp8KVCacheDataType kv_dt>
__global__ void convert_fp8_kernel(
  const Tin* __restrict__ src_cache,
  Tout* __restrict__ dst_cache,
  const float kv_scale,
  const int64_t block_stride) {
  const int64_t block_idx = blockIdx.x;
  for (int i = threadIdx.x; i < block_stride; i += blockDim.x) {
    int64_t idx = block_idx * block_stride + i;
    dst_cache[idx] = fp8::scaled_convert<Tout, Tin, kv_dt>(src_cache[idx], kv_scale);
  }
}

} // namespace vllm

#define CALL_CONVERT_FP8(Tout, Tin, KV_DTYPE)                                 \
  vllm::convert_fp8_kernel<Tout, Tin, KV_DTYPE><<<grid, block, 0, stream>>>(  \
    reinterpret_cast<Tin*>(src_cache.data_ptr()),                             \
    reinterpret_cast<Tout*>(dst_cache.data_ptr()),                            \
    kv_scale, \
    block_stride);

// Only for testing.
void convert_fp8(
  torch::Tensor& dst_cache,
  torch::Tensor& src_cache,
  const float kv_scale,
  const std::string& kv_cache_dtype)
{
  torch::Device src_device = src_cache.device();
  torch::Device dst_device = dst_cache.device();
  TORCH_CHECK(src_device.is_cuda(), "src must be on a GPU")
  TORCH_CHECK(dst_device.is_cuda(), "dst must be on a GPU")
  TORCH_CHECK(
    src_device.index() == dst_device.index(),
    "src and dst must be on the same GPU");
  at::cuda::OptionalCUDAGuard device_guard(src_device);

  int64_t num_blocks = src_cache.size(0);
  int64_t block_stride = src_cache.stride(0);

  dim3 grid(num_blocks);
  dim3 block(std::min(block_stride, int64_t(512)));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (kv_cache_dtype == "auto") {
    if (src_cache.dtype() == at::ScalarType::Float) {
      CALL_CONVERT_FP8(uint8_t, float, vllm::Fp8KVCacheDataType::kAuto);
    } else if (src_cache.dtype() == at::ScalarType::Half) {
      CALL_CONVERT_FP8(uint8_t, uint16_t, vllm::Fp8KVCacheDataType::kAuto);
    } else if (src_cache.dtype() == at::ScalarType::BFloat16) {
      CALL_CONVERT_FP8(uint8_t, __hip_bfloat16, vllm::Fp8KVCacheDataType::kAuto);
    } else if (dst_cache.dtype() == at::ScalarType::Float) {
      CALL_CONVERT_FP8(float, uint8_t, vllm::Fp8KVCacheDataType::kAuto);
    } else if (dst_cache.dtype() == at::ScalarType::Half) {
      CALL_CONVERT_FP8(uint16_t, uint8_t, vllm::Fp8KVCacheDataType::kAuto);
    } else if (dst_cache.dtype() == at::ScalarType::BFloat16) {
      CALL_CONVERT_FP8(__hip_bfloat16, uint8_t, vllm::Fp8KVCacheDataType::kAuto);
    }
  } else if (kv_cache_dtype == "fp8" || kv_cache_dtype == "fp8_e4m3") {
    if (src_cache.dtype() == at::ScalarType::Float) {
      CALL_CONVERT_FP8(uint8_t, float, vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (src_cache.dtype() == at::ScalarType::Half) {
      CALL_CONVERT_FP8(uint8_t, uint16_t, vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (src_cache.dtype() == at::ScalarType::BFloat16) {
      CALL_CONVERT_FP8(uint8_t, __hip_bfloat16, vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (dst_cache.dtype() == at::ScalarType::Float) {
      CALL_CONVERT_FP8(float, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (dst_cache.dtype() == at::ScalarType::Half) {
      CALL_CONVERT_FP8(uint16_t, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3);
    } else if (dst_cache.dtype() == at::ScalarType::BFloat16) {
      CALL_CONVERT_FP8(__hip_bfloat16, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3);
    }
  } else {
    TORCH_CHECK(false, "Unsupported data type: ", kv_cache_dtype);
  }
}
